
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void testKernel(float* A, unsigned int* B){
    *B = blockIdx.x;
}

void test(float* A_h){
    float* A_d;
    unsigned int* B_h = (unsigned int*)malloc(sizeof(unsigned int));
    unsigned int* B_d;

    hipMalloc((void**) &A_d, sizeof(unsigned int));
    hipMalloc((void**) &B_d, sizeof(unsigned int));
    
    hipMemcpy(A_d, A_h, sizeof(unsigned int), hipMemcpyHostToDevice);

    testKernel<<<3, sizeof(unsigned int)>>>(A_d, B_d);

    hipMemcpy(B_h, B_d, sizeof(unsigned int), hipMemcpyDeviceToHost);

    printf("Hello : %u", *B_h);

    hipFree(A_d);
    hipFree(B_d);
    free(B_h);
}

int main(int argc, char** argv){
    float* X = (float*) malloc(sizeof(unsigned int));

    test(X);

    free(X);
    return 0;
}
